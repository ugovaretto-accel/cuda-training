#include "hip/hip_runtime.h"
// #CSCS CUDA Training 
//
// #Example 1 - retrieve device info
//
// #Author Ugo Varetto
//
// #Goal: compute the maximum size for a 1D grid layout. i.e. the max size for 1D arrays that allows
//        to match a GPU thread with a single array element. 
//
// #Rationale: CUDA on arch < 2.x requires client code to configure the domain layout as a 1D or 2D grid of
//            1,2 or 3D blocks; it is not possible to simply set the GPU layout to match the
//            domain layout as is the case with OpenCL.
//
// #Solution: the max size for a 1D memory layout is computed as  (max num blocks per grid) x (max num threads per block)
//            i.e. min( deviceProp.maxThreadsDim[0], deviceProp.maxThreadsPerBlock  ) * deviceProp.maxGridSize[0]           
//
// #Code: finds number of devices and prints all the available information for each device,
//        relevant information is:
//          . deviceProp.maxGridSize[0] // max number of blocks in dimension zero
//          . deviceProp.maxThreadsDim[0] // max number of threads per block along dimesion 0
//          . deviceProp.maxThreadsPerBlock // max threads per block
//          . (optional) deviceProp.totalGlobalMem  //total amount of memory)     
//        proper code should perform some minimal error checking and iterate over
//        all the available devices
//        
// #Compilation: nvcc -arch=sm_13 1_device-query.cu -o device-query 
//
// #Execution: ./1_device-query
//
// #Note: by default the code prints all the information available for each graphics card;
//        #define MINIMAL to have the code print out only the relevant information        
//
// #Note: the code is C++ also because the default compilation mode for CUDA is C++, all functions
//        are named with C++ convention and the syntax is checked by default against C++ grammar rules 
//
// #Note: -arch=sm_13 allows the code to run on every card with hw architecture GT200 (gtx 2xx) or better


#include <iostream>
//#include <hip/hip_runtime.h> // automatically added by nvcc


int main( int argc, const char** argv) 
{
            
    int deviceCount = 0;
    if( hipGetDeviceCount( &deviceCount ) != hipSuccess ) {
        std::cout << "hipGetDeviceCount FAILED CUDA Driver and Runtime version may be mismatched.\n";
        std::cout << "\nFAILED\n";
        return 1;
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if ( deviceCount == 0 ) {
        std::cout << "There is no device supporting CUDA\n";
        return 1;
    }

    int dev = 0;
    int driverVersion = 0, runtimeVersion = 0;     
    for( dev = 0; dev != deviceCount; ++dev ) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties( &deviceProp, dev );
        if ( dev == 0) {
            // This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
            if( deviceProp.major == 9999 && deviceProp.minor == 9999 ) std::cout << "There is no device supporting CUDA.\n";
            else if (deviceCount == 1) std::cout << "There is 1 device supporting CUDA\n";
            else std::cout << "There are " << deviceCount << " devices supporting CUDA\n";
        }
        std::cout << "\nDevice" << dev << ": " << deviceProp.name << '\n';

    #ifndef MINIMAL        
        hipDriverGetVersion(&driverVersion);
        std::cout << "  CUDA Driver Version:                           " << driverVersion/1000 << '.' << driverVersion%100 << '\n';
        hipRuntimeGetVersion(&runtimeVersion);
        std::cout << "  CUDA Runtime Version:                          " << runtimeVersion/1000 << '.' << runtimeVersion%100 << '\n';
    
        std::cout << "  CUDA Capability Major/Minor version number:    " << deviceProp.major << '.' << deviceProp.minor << '\n';

        std::cout << "  Total amount of global memory:                 " << deviceProp.totalGlobalMem << " bytes\n";
        
        std::cout << "  Number of multiprocessors:                     " << deviceProp.multiProcessorCount << '\n';
            
        std::cout << "  Total amount of constant memory:               " << deviceProp.totalConstMem << " bytes\n";
        std::cout << "  Total amount of shared memory per block:       " << deviceProp.sharedMemPerBlock << " bytes\n";
        std::cout << "  Total number of registers available per block: " << deviceProp.regsPerBlock << '\n';
        std::cout << "  Warp size:                                     " << deviceProp.warpSize << '\n';
    #endif        
        std::cout << "  Maximum number of threads per block:           " << deviceProp.maxThreadsPerBlock << '\n';
        std::cout << "  Maximum sizes of each dimension of a block:    " 
                  << deviceProp.maxThreadsDim[0] << " x " 
                  << deviceProp.maxThreadsDim[1] << " x " 
                  << deviceProp.maxThreadsDim[2] << '\n';
        std::cout << "  Maximum sizes of each dimension of a grid:     " 
                  << deviceProp.maxGridSize[0] << " x " 
                  << deviceProp.maxGridSize[1] << " x "
                  << deviceProp.maxGridSize[2] << '\n';
    #ifndef MINIMAL
        std::cout << "  Maximum memory pitch:                          " << deviceProp.memPitch << " bytes\n";
        std::cout << "  Texture alignment:                             " << deviceProp.textureAlignment << " bytes\n";
        std::cout << "  Clock rate:                                    " << deviceProp.clockRate * 1e-6f << " GHz\n";
        std::cout << "  Concurrent copy and execution:                 " << (deviceProp.deviceOverlap ? "Yes" : "No") << '\n';
        std::cout << "  # of Asynchronous Copy Engines:                " << deviceProp.asyncEngineCount << '\n';
        std::cout << "  Run time limit on kernels:                     " << (deviceProp.kernelExecTimeoutEnabled ? "Yes\n" : "No\n");
        std::cout << "  Integrated:                                    " << (deviceProp.integrated ? "Yes\n" : "No\n");
        std::cout << "  Support host page-locked memory mapping:       " << (deviceProp.canMapHostMemory ? "Yes\n" : "No\n");
        std::cout << "  Compute mode:                                  " << (deviceProp.computeMode == hipComputeModeDefault ?
                                                                             "Default (multiple host threads can use this device simultaneously)\n" :
                                                                                  deviceProp.computeMode == hipComputeModeExclusive ?
                                                                                  "Exclusive (only one host thread at a time can use this device)\n" :
                                                                                  deviceProp.computeMode == hipComputeModeProhibited ?
                                                                                      "Prohibited (no host thread can use this device)\n" :
                                                                                      "Unknown\n");
        std::cout << "  Memory bus width:                              " << deviceProp.memoryBusWidth << " bytes\n";
        std::cout << "  Memory clock:                                  " << deviceProp.memoryClockRate * 1e-6f << " GHz\n";
    #endif
        std::cout << "  Concurrent kernel execution:                   " << (deviceProp.concurrentKernels ? "Yes\n" : "No\n");
        std::cout << "  Device has ECC support enabled:                " << (deviceProp.ECCEnabled ? "Yes\n" : "No\n");
        std::cout << "  Device is using TCC driver mode:               " << (deviceProp.tccDriver ? "Yes\n" : "No\n");
        std::cout << "  Unified addressing:                            " << (deviceProp.unifiedAddressing ? "Yes\n" : "No\n");
        std::cout << "  PCI bus id:                                    " << deviceProp.pciBusID << '\n';
        std::cout << "  PCI device id:                                 " << deviceProp.pciDeviceID << '\n';
   
    }

    return 0;
}
