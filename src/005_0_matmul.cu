// #CSCS CUDA Training 
//
// #Example 5 - (block) matrix-matrix multiply
//
// #Author: Ugo Varetto
//
// #Goal: multiply two matrices make use of shared memory to accelerate the computation
//
// #Rationale: shows how shared memory can be used to accelerate matrix-matrix operations   

// #Solution: copy matrix blocks into shared memory and perform matrix-matrix multiply
//            on shared memory buffers
//
// #Code: 1)  compute launch grid configuration
//        2)  allocate data on host(cpu) and device(gpu)
//        3)  initialize data directly on GPU
//        4)  read initialized data back from GPU so that we can use the same data on the CPU       
//        5)  create events
//        6)  issue time record request on start event
//        7)  launch kernel
//        8)  issue time record request on stop event
//        9)  synchronize stop event with end of kernel execution
//        10) read data back and print upper left corner of result matrix
//        11) perform computation on CPU and print upper left corner of result matrix
//        12) [optional] compare results; to avoid using a big eps (>=10^-4) use double precision 
//             
// #Compilation: nvcc -arch=sm_13 5_matmul.cu -o matmul
//
// #Execution: ./matmul
//
// #Note: kernel invocations ( foo<<<...>>>(...) ) are *always* asynchronous and a call to 
//        cudaThreadSynchronize() is required to wait for the end of kernel execution from
//        a host thread; in case of synchronous copy operations like cudaMemcpy(...,cudaDeviceToHost)
//        kernel execution is guaranteed to be terminated before data are copied
//
// #Note: -arch=sm_13 allows the code to run on every card with hw architecture GT200 (gtx 2xx) or better
//
// #Note: -arch=sm_13 is the lowest architecture version that supports double precision
//
// #Note: the example can be extended to read configuration data and matrix size from the command line
//
// #Note: try on both GT200 and GF100 architectures to verify the impact of L1 cache
 
//#include <cuda_runtime.h> // automatically added by nvcc

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>


typedef float real_t;

const size_t TILE_SIZE = 16;

// return matrix element given block and indices of element in block
__device__ real_t get_matrix_element( const real_t* m, //matrix
                                      int blockCol,    //column index of output block 
                                      int blockRow,    //row index of output row
                                      int col,         //local column index of block element
                                      int row,         //local row index of block element 
                                      int num_columns  //number of columns of matrix 'm'
                                     ) {                                           
  
    return m[ ( blockRow * blockDim.y + row ) * num_columns + blockCol * blockDim.x + col ];

}

// compute block matrix multiply:
// - matrix block size == TILE_SIZE == CUDA thread block size
// - grid (blocks x threads per block) matches the output matrix layout
// workflow: 
// 1) copy block from input matrices into local cache buffers
// 2) wait until all threads are done copying
// 3) identify output block location = C,R
// 4) iterate over blocks on row R in matrix 1 and blocks on column C in matrix 2;
//    for each block:
//    4.1) output element = c,r -> maps to current thread's x,y values
//    4.2) add to output element scalar product of row r in local cache 1 (matrix 1)
//         and column c in local cache 2 (matrix 2)
// 5) wait to perform next iteration until all block element have been computed        
//           
__global__ void block_matmul( const real_t* m1, const real_t* m2, real_t* mout,
                              int m1_columns, int m2_columns  ) { 
                                                                      
    __shared__ real_t M1[ TILE_SIZE ][ TILE_SIZE ];
    __shared__ real_t M2[ TILE_SIZE ][ TILE_SIZE ];     
        
    const int blockRow = blockIdx.y; 
    const int blockCol = blockIdx.x;
    const int row = threadIdx.y;
    const int col = threadIdx.x;
    real_t out = 0.f;
    for( int b = 0; b != m1_columns / TILE_SIZE; ++b ) {
          //copy data into shared memory
        M1[ row ][ col ] = get_matrix_element( m1, b, blockRow, col, row, m1_columns );
        M2[ row ][ col ] = get_matrix_element( m2, blockCol, b, col, row, m2_columns );
        __syncthreads(); // required to guarantee that data are computed before next step
                         // where a thread accesses data computed by other threads
        for( int k = 0; k != TILE_SIZE; ++k ) {
            out += M1[ row ][ k ] * M2[ k ][ col ];           
        }
        __syncthreads(); // required to avoid that some threads start modifying
                         // data in cache before all threads have exited for loop    
    }
    mout[ ( blockRow * blockDim.y + row ) * m2_columns + blockCol * blockDim.x + col ] = out;     
}

// simple matrix multiplication; grid layout matches output matrix; note that
// although this method is slower than the block multiply, it is still much faster
// than running on the cpu
__global__ void matmul( const real_t* m1, const real_t* m2, real_t* mout,
                        int m1_columns, int m2_columns  ) { // m1_columns == m2_rows
                                                            // mout = m1_rows x m2_columns
    const int row = blockIdx.y * blockDim.y + threadIdx.y; 
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    real_t out = 0.f;//m1[ row * m1_columns + 0 ] * m2[ 0 * m2_columns + col ];

    for( int k = 0; k != m1_columns; ++k ) {
        out += m1[ row * m1_columns + k ] * m2[ k * m2_columns + col ];
    }
    mout[ row * m2_columns + col ] = out;
}

__global__ void init_matrix( real_t* m ) {
    const int c = threadIdx.x + blockDim.x * blockIdx.x;
    const int r = threadIdx.y + blockDim.y * blockIdx.y;
    const int idx = c + gridDim.x * blockDim.x * r; 
    const real_t s = gridDim.x * gridDim.y;
    m[ idx ] = real_t( idx ) / s; 
}

// standard matrix-matrix multiply
void matmul_ref( const real_t* m1, const real_t* m2, real_t* mout,
                 int m1_rows, int m1_columns, int m2_columns  ) {
                     
    for( int row = 0; row != m1_rows; ++row ) {
        for( int col = 0; col != m2_columns; ++col ) {
            mout[ row * m2_columns + col ] = 0.f; 
            for( int k = 0; k != m1_columns; ++k ) {
                mout[ row * m2_columns + col ] += m1[ row * m1_columns + k ] * m2[ k * m2_columns + col ];
            }
        }
    }
}

// compare floating point arrays
bool compare( const real_t* v1, const real_t* v2, size_t N, real_t eps ) { 
    for( int i = 0; i != N; ++i ) {
        if( std::fabs( v1[ i ] - v2[ i ] ) > eps ) return false;
    }
    return true;
}

// print matrix; 'stride' in case we want to print only a subset
// of the matrix: in this case c != stride
void print_matrix( const real_t* m, int r, int c, int stride ) {
    for( int i = 0; i != r; ++i ) {
        for( int j = 0; j != c; ++j ) std::cout << m[ i * stride + j ] << ' ';
        std::cout << '\n';
    }
    std::cout << std::endl;   
}

//------------------------------------------------------------------------------
int main(int argc, char** argv ) {
    
    //1024 x 1024 matrices
    const dim3 BLOCKS( 64, 64 );
    const dim3 THREADS_PER_BLOCK( 16, 16 ); 
    const int ROWS = BLOCKS.y * THREADS_PER_BLOCK.y;
    const int COLUMNS =  BLOCKS.x * THREADS_PER_BLOCK.x;
    const size_t ARRAY_SIZE = ROWS * COLUMNS;
    const size_t BYTE_SIZE = ARRAY_SIZE * sizeof( real_t );
      
    // device storage for gpu computation
    real_t* dev_m1 = 0;
    real_t* dev_m2 = 0;
    real_t* dev_mout = 0;
    hipMalloc( &dev_m1,  BYTE_SIZE );
    hipMalloc( &dev_m2,  BYTE_SIZE );
    hipMalloc( &dev_mout, BYTE_SIZE );
    //host storage for reading the output of gpu computation
    std::vector< real_t> host_mout( ARRAY_SIZE );
    
    // host storage for cpu computation
    std::vector< real_t > m1( ARRAY_SIZE );
    std::vector< real_t > m2( ARRAY_SIZE );
    std::vector< real_t > mout( ARRAY_SIZE );

    // initialize matrix with kernel; much faster than using
    // for loops on the cpu
    init_matrix<<<dim3( COLUMNS, ROWS ), 1>>>( dev_m1 );
    init_matrix<<<dim3( COLUMNS, ROWS ), 1>>>( dev_m2 );
 
    // copy initialized data into host arrays for further processing on the gpu
    hipMemcpy( &m1[ 0 ], dev_m1, BYTE_SIZE, hipMemcpyDeviceToHost );
    hipMemcpy( &m2[ 0 ], dev_m2, BYTE_SIZE, hipMemcpyDeviceToHost );
   
    // print upper 4x4 left corner of input matrix 1
    std::cout << "INPUT MATRIX 1 - " << ROWS << " rows, " << COLUMNS << " columns" << std::endl;
    print_matrix( &m1[ 0 ], 4, 4, COLUMNS );
    // print upper 4x4 left corner of input matrix 2
    std::cout << "INPUT MATRIX 2 - " << ROWS << " rows, " << COLUMNS << " columns" << std::endl;
    print_matrix( &m2[ 0 ], 4, 4, COLUMNS );
    
    // create events for timing execution
    hipEvent_t start = hipEvent_t();
    hipEvent_t stop  = hipEvent_t();
    hipEventCreate( &start );
    hipEventCreate( &stop  );
    
    // record time into start event 
    hipEventRecord( start, 0 ); // 0 is the default stream id

#ifdef BLOCK_MULTIPLY    
    // execute kernel
    block_matmul<<<BLOCKS, THREADS_PER_BLOCK>>>( dev_m1, dev_m2, dev_mout,  COLUMNS, COLUMNS );
#else  
    matmul<<<BLOCKS, THREADS_PER_BLOCK>>>( dev_m1, dev_m2, dev_mout,  COLUMNS, COLUMNS );
#endif  
  
    // issue request to record time into stop event
    hipEventRecord( stop, 0 );
    // synchronize stop event to wait for end of kernel execution on stream 0
    hipEventSynchronize( stop );
    // compute elapsed time (done by CUDA run-time) 
    float elapsed = 0.f;
    hipEventElapsedTime( &elapsed, start, stop );
    
    std::cout << "Elapsed time (ms): " << elapsed << std::endl;

    // copy output data from device(gpu) to host(cpu)
    hipMemcpy( &host_mout[ 0 ], dev_mout, BYTE_SIZE, hipMemcpyDeviceToHost );

    // print upper 4x4 corner of output matrix
    std::cout << "\nGPU OUTPUT MATRIX - " << ROWS << " rows, " << COLUMNS << " columns" << std::endl;
    print_matrix( &host_mout[ 0 ], 4, 4, COLUMNS );

    // compute on cpu
    matmul_ref( &m1[ 0 ], &m2[ 0 ], &mout[ 0 ], ROWS, COLUMNS, COLUMNS );  
    // print upper 4x4 corner of output matrix
    std::cout << "\nCPU OUTPUT MATRIX - " << ROWS << " rows, " << COLUMNS << " columns" << std::endl;
    print_matrix( &mout[ 0 ], 4, 4, COLUMNS );

#ifdef COMPARE_RESULTS
    // warning: requires real_t = double to pass
    std::cout << "Comparing... ";
    if( compare( &host_mout[ 0 ], &mout[ 0 ], ARRAY_SIZE, 0.00001 ) ) std::cout << "PASSED" << std::endl;
    else std::cout << "FAILED" << std::endl;
#endif

    // free memory
    hipFree( dev_m1 );
    hipFree( dev_m2 );
    hipFree( dev_mout );

    // release events
    hipEventDestroy( start );
    hipEventDestroy( stop  );

    return 0;
}


