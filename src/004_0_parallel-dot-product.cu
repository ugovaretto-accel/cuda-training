// #CUDA Training
//
// #Example 4 - dot product, pattial dot products on GPU, final reduction on CPU
//
// #Author: Ugo Varetto
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <vector>

typedef float real_t;

const size_t BLOCK_SIZE = 16;


//Reduce at each step by summing in place element[i] + element[i+step]
//where 'step'starts at half array length and gets divided by two at each step
//   _______
//  v       v
//1 1 1 1 1 1 1 1
//^-------^
//
//2 2 2 2
//4 4
//8 <-- return reduced value back to host code
//launch grid size <= array size
//number of returned partial redutions == number of blocks
__global__ void partial_dot(const real_t* v1, const real_t* v2, real_t* out,
                            int N) {
    __shared__ real_t cache[BLOCK_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cache[threadIdx.x] = 0.f;
    while (i < N) {
        cache[threadIdx.x] += v1[i] * v2[i];
        i += gridDim.x * blockDim.x;
    }
    __syncthreads();  // required because later on the current thread is
                      // accessing data written by another thread
    i = BLOCK_SIZE / 2;
    while (i > 0) {
        if (threadIdx.x < i) cache[threadIdx.x] += cache[threadIdx.x + i];
        __syncthreads();
        i /= 2;  // not sure bitwise operations are actually faster
    }

    if (threadIdx.x == 0) out[blockIdx.x] = cache[0];
}

real_t dot(const real_t* v1, const real_t* v2, int N) {
    real_t s = 0;
    for (int i = 0; i != N; ++i) {
        s += v1[i] * v2[i];
    }
    return s;
}

__global__ void init_vector(real_t* v, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < N) {
        v[i] = 1.0f;  // real_t( i ) / 1000000.f;
        i += gridDim.x * blockDim.x;
    }
}

//------------------------------------------------------------------------------
int main(int argc, char** argv) {
    const size_t ARRAY_SIZE = 1024;  // 1024 * 1024; //1Mi elements
    const int BLOCKS = 64;           // 512;
    const int THREADS_PER_BLOCK =
        BLOCK_SIZE;  // 256; // total threads = 512 x 256 = 128ki threads; //
                     // each thread spans 8 array elements
    const size_t SIZE = ARRAY_SIZE * sizeof(real_t);

    // device storage
    real_t* dev_v1 = 0;    // vector 1
    real_t* dev_v2 = 0;    // vector 2
    real_t* dev_vout = 0;  // partial redution = number of blocks
    hipMalloc(&dev_v1, SIZE);
    hipMalloc(&dev_v2, SIZE);
    hipMalloc(&dev_vout, BLOCKS * sizeof(real_t));

    // host storage
    std::vector<real_t> host_v1(ARRAY_SIZE);
    std::vector<real_t> host_v2(ARRAY_SIZE);
    std::vector<real_t> host_vout(BLOCKS);

    // initialize vector 1 with kernel; much faster than using for loops on the
    // cpu
    init_vector<<<1024, 256>>>(dev_v1, ARRAY_SIZE);
    hipMemcpy(&host_v1[0], dev_v1, SIZE, hipMemcpyDeviceToHost);
    // initialize vector 2 with kernel; much faster than using for loops on the
    // cpu
    init_vector<<<1024, 256>>>(dev_v2, ARRAY_SIZE);
    hipMemcpy(&host_v2[0], dev_v2, SIZE, hipMemcpyDeviceToHost);

    // execute kernel
    partial_dot<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_v1, dev_v2, dev_vout,
                                               ARRAY_SIZE);

    // copy output data from device(gpu) to host(cpu)
    hipMemcpy(&host_vout[0], dev_vout, BLOCKS * sizeof(real_t),
               hipMemcpyDeviceToHost);

    // print dot product by summing up the partially reduced vectors
    std::cout << "GPU: "
              << std::accumulate(host_vout.begin(), host_vout.end(), real_t(0))
              << std::endl;

    // print dot product on cpu
    std::cout << "CPU: " << dot(&host_v1[0], &host_v2[0], ARRAY_SIZE)
              << std::endl;

    // free memory
    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_vout);

    return 0;
}
