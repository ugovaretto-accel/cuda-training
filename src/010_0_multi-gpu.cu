#include "hip/hip_runtime.h"
// #CSCS CUDA Training 
//
// #Example 10 - CUDA 4, peer to peer access, parallel execution on separate GPUs
//
// #Author Ugo Varetto
//
// #Goal: run kernels on separate GPUs passing the same pointer to both kernels; have
//        each kernel operate on a subset of the data
//
// #Rationale: shows how the same memory can be accessed from kernels in separate GPUs and
//             how to time the concurrent execution of kernels
//
// #Solution: use setCudaDevice and cudaEnablePeerAccess to select device and
//            enable sharing of memory
//
// #Code: 1) allocate device memory
//        2) select first GPU
//        3) launch kernel
//        4) copy data back from GPU 
//        5) select second GPU
//        6) launch other kernel
//        7) copy data back from GPU 
//        8) free memory
//        
// #Compilation: nvcc -arch=sm_20 10_multi-gpu.cu -o multi-gpu
//
// #Execution: ./multi-gpu
//
// #Note: Fermi (2.0) or better required; must be compiled with sm_2x
//
// #Note: Requires at least two GPUs
//
// #Note: timing execution of separate parallel kernels requires separate events to be created and used
//        in the context associated with each device i.e. invoke setDevice() before performing operations
//        on events
//
// #Note: kernel invocations ( foo<<<...>>>(...) ) are *always* asynchronous and a call to 
//        hipDeviceSynchronize() is required to wait for the end of kernel execution from
//        a host thread; in case of synchronous copy operations like hipMemcpy(...,cudaDeviceToHost)
//        kernel execution is guaranteed to be terminated before data are copied
//
// #Note: try to change the grid size and check how this affects performance 

//#include <hip/hip_runtime.h> // automatically added by nvcc
#include <iostream>
#include <vector>
#include <string>
#include <ctime>
#include <algorithm>
#include <cmath>

typedef double real_t;


__device__ size_t get_global_index( const dim3& gridSize,
                                    const dim3& offset ) {
    const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t z = blockIdx.z * blockDim.z + threadIdx.z;
    const size_t yStride = gridSize.x;
    const size_t zStride = yStride * gridSize.y;
    return  ( z + offset.z ) * zStride + ( y + offset.y ) * yStride + x + offset.x;
}


__global__ void kernel_on_dev1( real_t* buffer, dim3 gridSize, dim3 offset ) {
    buffer[ get_global_index( gridSize, offset ) ] =   2.0;  
}

__global__ void kernel_on_dev2( real_t* buffer, dim3 gridSize, dim3 offset ) {
    buffer[ get_global_index( gridSize, offset ) ] =  -2.0;  
}

__global__ void init( real_t* buffer, dim3 gridSize, dim3 offset ) {
    buffer[ get_global_index( gridSize, offset ) ] =   1.0f;
}


void print_ptr_attr( const hipPointerAttribute_t& pa ) {
    std::cout << "\nPointer attributes:\n";
    std::string mt = pa.memoryType == hipMemoryTypeHost ? "hipMemoryTypeHost"
                                                         : "hipMemoryTypeDevice";
    std::cout << "  memoryType:    " << mt << std::endl;
    std::cout << "  device:        " << pa.device << std::endl;
    std::cout << "  devicePointer: " << std::hex << pa.devicePointer << std::endl;
    std::cout << "  hostPointer:   " << pa.hostPointer << std::endl;
}

//------------------------------------------------------------------------------
int main( int , char**  ) {
    
    real_t* dev_buffer = 0;
    const size_t SZ = 512;
    const size_t SIZE = SZ * SZ * SZ;
    const size_t BYTE_SIZE = SIZE * sizeof( real_t );
    int ndev = 0;
    hipGetDeviceCount( &ndev );
    if( ndev < 2 ) {
        std::cout << "At least two GPU devices required, " << ndev << " found" << std::endl;
        return 1;
    }
   
    // check if possible to access device 0 from device 1
    int yes = 0;
    int client_device = 1; // device willing to acces data on foreign device
    int host_device = 0;   // device on which data have been allocated
    hipDeviceCanAccessPeer( &yes, client_device, host_device );
    if( yes != 1 ) {
        std::cout << "Cannot access " << host_device << " from device " << client_device << std::endl;
        return 1;
    }


    std::cout << "\nGrid size: " << BYTE_SIZE / double( 1024 * 1024 * 1024 ) << " GiB" << std::endl;
       
    // on device 0
    hipSetDevice( 0 );
    // allocate and initialize data
    hipMalloc( &dev_buffer, BYTE_SIZE );
    init<<< dim3( SZ, SZ, SZ ), 1 >>>( dev_buffer, dim3( SZ, SZ, SZ ), dim3( 0, 0, 0 ) );
    hipDeviceSynchronize(); 
    hipPointerAttribute_t pointer_attr;
    // print pointer attributes
    hipPointerGetAttributes( &pointer_attr, dev_buffer );
    print_ptr_attr( pointer_attr );
    // create events for timing 
    hipEvent_t init_start, init_stop;
    hipEventCreate( &init_start );
    hipEventCreate( &init_stop  );
    hipEventRecord( init_start, 0 );
    // launch kernel on half domain and time execution *before* sharing memory
    kernel_on_dev1<<< dim3( SZ, SZ, SZ / 2 ), 1 >>>( dev_buffer, dim3( SZ, SZ, SZ ), dim3( 0, 0, 0 ) ); 
    hipEventRecord( init_stop, 0 );
    hipEventSynchronize( init_stop );
    hipDeviceSynchronize();
    float elapsed_half_no_sharing;
    hipEventElapsedTime( &elapsed_half_no_sharing, init_start, init_stop );
    std::cout << "\nKernel on first device on half domain before sharing:  "
              << elapsed_half_no_sharing << " ms\n" << std::endl;
    hipEventRecord( init_start, 0 );
    // launch kernel on entire grid and time execution
    kernel_on_dev1<<< dim3( SZ, SZ, SZ ), 1 >>>( dev_buffer, dim3( SZ, SZ, SZ ), dim3( 0, 0, 0 ) ); 
    hipEventRecord( init_stop, 0 );
    hipEventSynchronize( init_stop );
    hipDeviceSynchronize();
    float elapsed_full_no_sharing;
    hipEventElapsedTime( &elapsed_full_no_sharing, init_start, init_stop );
    std::cout << "\nKernel on first device on full domain before sharing:  " 
              << elapsed_full_no_sharing << " ms\n" << std::endl;
 
    // switch to device 1
    hipSetDevice( 1 );
    // print again pointer attributes *before* sharing data 
    std::cout << "Before hipDeviceEnablePeerAccess:" << std::endl;
    hipPointerGetAttributes( &pointer_attr, dev_buffer );
    print_ptr_attr( pointer_attr );
    // enable sharing with device 0
    hipDeviceEnablePeerAccess( 0, 0 );
    // print pointer attributes *after* enabling sharing of data
    hipPointerGetAttributes( &pointer_attr, dev_buffer );
    print_ptr_attr( pointer_attr );
    std::cout << "After hipDeviceEnablePeerAccess:"  << std::endl;
    print_ptr_attr( pointer_attr );
    
    hipSetDevice( 0 );
    // launch kernel on front part of domain
    hipEvent_t start1, stop1, start12, stop12, start2, stop2;
    hipEventCreate( &start1 );
    hipEventCreate( &start12 );
    hipEventCreate( &stop1  );
    hipEventCreate( &stop12  );
    hipSetDevice( 1 );
    hipEventCreate( &start2 );
    hipEventCreate( &stop2  );
    hipSetDevice( 0 );
    hipEventRecord( start1, 0 );
    clock_t cpu_start = clock();    
    kernel_on_dev1<<< dim3( SZ, SZ, SZ / 2 ), 1 >>>( dev_buffer, dim3( SZ, SZ, SZ ), dim3( 0, 0, 0 ) );
    hipSetDevice( 1 );
    // launch kernel on back part of domain
    hipEventRecord( start2, 0 );
    kernel_on_dev2<<< dim3( SZ, SZ, SZ / 2 ), 1 >>>( dev_buffer, dim3( SZ, SZ, SZ ), dim3( 0, 0, SZ / 2 ) );
    hipEventRecord( stop2, 0 );
    hipEventSynchronize( stop2 );
    hipDeviceSynchronize();
    hipSetDevice( 0 );
    hipEventRecord( stop1, 0 );
    hipEventSynchronize( stop1 );
    hipDeviceSynchronize();
    clock_t cpu_end = clock();

    
    // on POSIX systems CLOCKS_PER_SECOND is always 1E6
    std::cout << "\nCPU time: " << double( cpu_end - cpu_start ) / 1E3 << " ms"<< std::endl;
   
    float e1, e2;
    hipEventElapsedTime( &e1, start1, stop1 );
    hipSetDevice( 1 );
    hipEventElapsedTime( &e2, start2, stop2 );
    hipSetDevice( 0 );
    std::cout << "GPU time: " <<  std::max( e1, e2 ) << " ms\n" << std::endl;      
    
    std::vector< real_t > host_buffer( SIZE );
    hipMemcpy( &host_buffer[ 0 ], dev_buffer, BYTE_SIZE, hipMemcpyDeviceToHost );
    std::cout << ": " << host_buffer.front() << "..." << host_buffer.back() << std::endl; 
    
    std::cout << "Half domain: exec. time without sharing / exec. time with sharing: " 
              << elapsed_half_no_sharing / std::max( e1, e2 ) << std::endl;
    std::cout << "Full domain: exec. time without sharing / exec. time with sharing: " 
              << elapsed_full_no_sharing / std::max( e1, e2 ) << std::endl;
    std::cout << "Gain: " << std::ceil( 100 * ( elapsed_full_no_sharing / std::max( e1, e2 ) - 1 ) ) << '%' << std::endl;
    
    // disable peer access and re-run first kernel to verify that results are consistent
    hipSetDevice( 1 );
    hipDeviceDisablePeerAccess( 0 );
    hipSetDevice( 0 ); 
    hipEventRecord( init_start, 0 );
    // launch kernel on half domain and time execution *before* sharing memory
    kernel_on_dev1<<< dim3( SZ, SZ, SZ / 2 ), 1 >>>( dev_buffer, dim3( SZ, SZ, SZ ), dim3( 0, 0, 0 ) ); 
    hipEventRecord( init_stop, 0 );
    hipEventSynchronize( init_stop );
    hipDeviceSynchronize();
    float elapsed_half_sharing_disable;
    hipEventElapsedTime( &elapsed_half_sharing_disable, init_start, init_stop );
    std::cout << "\nKernel on first device on half domain after disabling sharing:  "
              << elapsed_half_sharing_disable << " ms\n" << std::endl;
    
    hipEventDestroy( init_start );
    hipEventDestroy( init_stop  );
    hipEventDestroy( start1 );
    hipEventDestroy( stop1  );
    hipFree( dev_buffer );
    hipSetDevice( 1 );
    hipEventDestroy( start2 );
    hipEventDestroy( stop2  );
    
    
    return 0;
}
