// #CUDA Training
//
// #Example 2.1 - sum vectors, launch grid size < array size
//
// #Author Ugo Varetto
//

//#include <cuda_runtime.h> // automatically added by nvcc

#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

typedef float real_t;

// In this case the number of GPU threads is smaller than the number of elements
// in the domain: every thread iterates over multple elements to ensure than the
// entire domain is covered
__global__ void sum_vectors(const real_t* v1, const real_t* v2, real_t* out,
                            size_t num_elements) {
    // compute current thread id
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    // iterate over vector: grid can be smaller than vector, it is therefore
    // required that each thread iterate over more than one vector element
    while (xIndex < num_elements) {
        out[xIndex] = v1[xIndex] + v2[xIndex];
        xIndex += gridDim.x * blockDim.x;
    }
}

//------------------------------------------------------------------------------
int main(int, char**) {
    const int VECTOR_SIZE = 0x10000 + 1;  // vector size 65537
    const int MAX_NUMBER_OF_THREADS = VECTOR_SIZE / 5;
    const int SIZE = sizeof(real_t) * VECTOR_SIZE;  // total size in bytes
    const int THREADS_PER_BLOCK = 32;  // number of gpu threads per block
    const int NUMBER_OF_BLOCKS = MAX_NUMBER_OF_THREADS / THREADS_PER_BLOCK;

    // host allocated storage; use std vectors to simplify memory management
    // and initialization
    std::vector<real_t> v1(VECTOR_SIZE, 1.f);    // initialize all elements to 1
    std::vector<real_t> v2(VECTOR_SIZE, 2.f);    // initialize all elements to 2
    std::vector<real_t> vout(VECTOR_SIZE, 0.f);  // initialize all elements to 0

    // gpu allocated storage
    real_t* dev_in1 = 0;  // vector 1
    real_t* dev_in2 = 0;  // vector 2
    real_t* dev_out = 0;  // result value
    hipMalloc(&dev_in1, SIZE);
    hipMalloc(&dev_in2, SIZE);
    hipMalloc(&dev_out, SIZE);

    // copy data to GPU
    hipMemcpy(dev_in1, &v1[0], SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_in2, &v2[0], SIZE, hipMemcpyHostToDevice);

    // execute kernel with num threads >= num elements
    sum_vectors<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(dev_in1, dev_in2,
                                                         dev_out, VECTOR_SIZE);

    // read back result
    hipMemcpy(&vout[0], dev_out, SIZE, hipMemcpyDeviceToHost);

    // print first and last element of vector
    std::cout << "result: " << vout.front() << ".." << vout.back() << std::endl;

    // free memory
    hipFree(dev_in1);
    hipFree(dev_in2);
    hipFree(dev_out);

    return 0;
}
