// #CUDA Training
//
// #Example 4.3 - dot product with two step reduction, all processing on GPU
//
// #Author: Ugo Varetto
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

typedef float real_t;

const size_t BLOCK_SIZE = 16;

//------------------------------------------------------------------------------

// Full on-gpu reduction

// each block atomically increments this variable when done
// performing the first reduction step
__device__ unsigned int count = 0;
// shared memory used by partial_dot and sum functions
// for temporary partial reductions; declare as global variable
// because used in more than one function
__shared__ real_t cache[BLOCK_SIZE];

// partial dot product: each thread block produces a single value
__device__ void partial_dot(const real_t* v1, const real_t* v2, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    cache[threadIdx.x] = 0.f;
    // the threads in the thread block iterate over the entire domain; iteration
    // happens whenever the total number of threads is lower than the domain
    // size
    while (i < N) {
        cache[threadIdx.x] += v1[i] * v2[i];
        i += gridDim.x * blockDim.x;
    }
    __syncthreads();  // required because later on the current thread is
                      // accessing data written by another thread
    i = BLOCK_SIZE / 2;
    while (i > 0) {
        if (threadIdx.x < i) cache[threadIdx.x] += cache[threadIdx.x + i];
        __syncthreads();
        i /= 2;
    }
}

// perform parallel dot product in two steps:
// 1) each block computes a single value and stores it into an array of size ==
// number of blocks 2) the last block to finish step (1) performs a reduction on
// the array produced in the above steps parameters: v1 first input vector v2
// second input vector N  size of input vector out output vector: size MUST be
// equal to the number of GPU blocks since it us used
//     for partial reduction; result is at position 0
__global__ void full_dot(const real_t* v1, const real_t* v2, int N,
                         real_t* out) {
    // true if last block to compute value
    bool lastBlock = false; 
    // each block computes a value
    partial_dot(v1, v2, N);
    if (threadIdx.x == 0) {
        // value is stored into output array by first thread of each block
        out[blockIdx.x] = cache[0];
        // wait for value to be available to all the threads on the device
        __threadfence();
        // increment atomic counter and retrieve value
        const unsigned int v = atomicInc(&count, gridDim.x);
        // check if last block to perform computation
        lastBlock = (v == gridDim.x - 1);
    }
    
    // last block performs a the final reduction steps which produces one single
    // value
    if (lastBlock && threadIdx.x == 0) {
        float r = 0;
        for(float* b = out; b != out + gridDim.x; r += *b++);
        out[0] = r;
    }
}

//------------------------------------------------------------------------------

// cpu implementation of dot product
real_t dot(const real_t* v1, const real_t* v2, int N) {
    real_t s = 0;
    for (int i = 0; i != N; ++i) {
        s += v1[i] * v2[i];
    }
    return s;
}

// initialization function run on the GPU
__global__ void init_vector(real_t* v, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < N) {
        v[i] = 1.0f;  // real_t( i ) / 1000000.f;
        i += gridDim.x * blockDim.x;
    }
}

//------------------------------------------------------------------------------
int main(int argc, char** argv) {
    const size_t ARRAY_SIZE = 1024;  // 1024 * 1024; //1Mi elements
    const int BLOCKS = 64;           // 512;
    const int THREADS_PER_BLOCK =
        BLOCK_SIZE;  // 256; // total threads = 512 x 256 = 128ki threads;
    const size_t SIZE = ARRAY_SIZE * sizeof(real_t);

    // device storage
    real_t* dev_v1 = 0;   // vector 1
    real_t* dev_v2 = 0;   // vector 2
    real_t* dev_out = 0;  // result array, final result is at position 0;
                          // also used for temporary GPU storage,
                          // must have size == number of thread blocks
    hipMalloc(&dev_v1, SIZE);
    hipMalloc(&dev_v2, SIZE);
    hipMalloc(&dev_out, sizeof(real_t) * BLOCKS);

    // host storage
    std::vector<real_t> host_v1(ARRAY_SIZE);
    std::vector<real_t> host_v2(ARRAY_SIZE);
    real_t host_out = 0.f;

    // initialize vector 1 with kernel; much faster than using for loops on the
    // cpu
    init_vector<<<1024, 256>>>(dev_v1, ARRAY_SIZE);
    hipMemcpy(&host_v1[0], dev_v1, SIZE, hipMemcpyDeviceToHost);
    // initialize vector 2 with kernel; much faster than using for loops on the
    // cpu
    init_vector<<<1024, 256>>>(dev_v2, ARRAY_SIZE);
    hipMemcpy(&host_v2[0], dev_v2, SIZE, hipMemcpyDeviceToHost);

    // execute kernel
    full_dot<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_v1, dev_v2, ARRAY_SIZE,
                                            dev_out);
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

    // copy output data from device(gpu) to host(cpu)
    hipMemcpy(&host_out, dev_out, sizeof(real_t), hipMemcpyDeviceToHost);

    // print dot product by summing up the partially reduced vectors
    std::cout << "GPU: " << host_out << std::endl;

    // print dot product on cpu
    std::cout << "CPU: " << dot(&host_v1[0], &host_v2[0], ARRAY_SIZE)
              << std::endl;

    // free memory
    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_out);

    return 0;
}
