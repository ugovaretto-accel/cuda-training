// #CUDA Training
//
// #Example 2.0 - sum vectors, launch gid size == array size
//
// #Author Ugo Varetto
//


//#include <cuda_runtime.h> // automatically added by nvcc

#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

typedef float real_t;

// In this case the kernel assumes that the computation was started with enough
// threads to cover the entire domain. This is the preferred solution provided
// there are enough threads to cover the entire domain which might not be the
// case in case of a 1D grid layout (max number of threads = 512 threads per
// block x 65536  blocks = 2^25 = 32 Mi threads)
__global__ void sum_vectors(const real_t* v1, const real_t* v2, real_t* out,
                            size_t num_elements) {
    // compute current thread id
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    // since we assume that num threads >= num element we need to make sure we
    // do not write outside the range of the output buffer
    if (xIndex < num_elements) out[xIndex] = v1[xIndex] + v2[xIndex];
}

//------------------------------------------------------------------------------
int main(int, char**) {
    const int VECTOR_SIZE = 0x10000 + 1;            // vector size 65537
    const int SIZE = sizeof(real_t) * VECTOR_SIZE;  // total size in bytes
    const int THREADS_PER_BLOCK = 32;  // number of gpu threads per block

    // block size: the number of threads per block multiplied by the number of
    // blocks must be at least equal to NUMBER_OF_THREADS
    const int NUMBER_OF_BLOCKS =
        (VECTOR_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    // if number of threads is not evenly divisable by the number of threads per
    // block we need an additional block; the above code can be rewritten as if(
    // NUMBER_OF_THREADS % THREADS_PER_BLOCK == 0) BLOCK_SIZE =
    // NUMBER_OF_THREADS / THREADS_PER_BLOCK; else BLOCK_SIZE =
    // NUMBER_OF_THREADS / THREADS_PER_BLOCK + 1

    // host allocated storage; use std vectors to simplify memory management
    // and initialization
    std::vector<real_t> v1(VECTOR_SIZE, 1.f);    // initialize all elements to 1
    std::vector<real_t> v2(VECTOR_SIZE, 2.f);    // initialize all elements to 2
    std::vector<real_t> vout(VECTOR_SIZE, 0.f);  // initialize all elements to 0

    // gpu allocated storage
    real_t* dev_in1 = 0;  // vector 1
    real_t* dev_in2 = 0;  // vector 2
    real_t* dev_out = 0;  // result value
    hipMalloc(&dev_in1, SIZE);

    hipMalloc(&dev_in2, SIZE);
    hipMalloc(&dev_out, SIZE);

    // copy data to GPU
    hipMemcpy(dev_in1, &v1[0], SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_in2, &v2[0], SIZE, hipMemcpyHostToDevice);

    // execute kernel with num threads >= num elements
    sum_vectors<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(dev_in1, dev_in2,
                                                         dev_out, VECTOR_SIZE);
    // read back result
    hipMemcpy(&vout[0], dev_out, SIZE, hipMemcpyDeviceToHost);

    // print first and last element of vector
    std::cout << "result: " << vout.front() << ".." << vout.back() << std::endl;

    // free memory
    hipFree(dev_in1);
    hipFree(dev_in2);
    hipFree(dev_out);

    return 0;
}
