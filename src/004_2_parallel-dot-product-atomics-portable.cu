// #CUDA Training
//
// #Example 4.2 - dot product with CAS/Exch- generic version with custom mutex,
//                originally named "portable" because at the time floating
//                point atomics where not implemented on all available
//                generations of GPUs.
//
// #Author: Ugo Varetto
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

typedef float real_t;

const size_t BLOCK_SIZE = 16;

// lock mutex: atomicCAS sets variable to third argument if
// variable value is equal to second argument and returns previous value.
// In this case we spin in the while loop until the mutex is set to 1 i.e.
// until its return value is != 0 and exit the loop only after the mutex has
// been acquired i.e. has been set to 1 after the mutex has been released i.e.
// set to zero.
__device__ void lock(int* mutex) {
    while (atomicCAS(mutex, 0, 1) != 0)
        ;
}

// set mutex to zero; note that we do not need to use an atomic op here;
// it is however preferred to access the same memory accessed by atomic
// functions only with atomic functions for consistency reasons: atomic
// transations and regular memory access follow different paths on the GPU; it
// might *appear* that the unlock doesn't look in sync with the lock, although
// the final result will be correct.
__device__ void unlock(int* mutex) { atomicExch(mutex, 0); }

// custom implementation of atomic add for floating point variables
__device__ void atomicAddF(real_t* pv, real_t v, int* mutex) {
    lock(mutex);
    *pv += v;
    unlock(mutex);
}

// dot product entirely executed on the GPU;
// last reduction step is executed by serializing
// access to the output variable through a mutex
// Reduce at each step by summing in place element[i] + element[i+step]
// where 'step'starts at half array length and gets divided by two at each step
//   _______
//  v       v
// 1 1 1 1 1 1 1 1
//^-------^
//
// 2 2 2 2
// 4 4
// 8 <-- add value to global result
__global__ void full_dot(const real_t* v1, const real_t* v2, real_t* out, int N,
                         int* mutex) {
    __shared__ real_t cache[BLOCK_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cache[threadIdx.x] = 0.f;
    while (i < N) {
        cache[threadIdx.x] += v1[i] * v2[i];
        i += gridDim.x * blockDim.x;
    }
    __syncthreads();  // required because later on the current thread is
                      // accessing data written by another thread
    i = BLOCK_SIZE / 2;
    while (i > 0) {
        if (threadIdx.x < i) cache[threadIdx.x] += cache[threadIdx.x + i];
        __syncthreads();
        i /= 2;  // not sure bitwise operations are actually faster
    }
#ifndef NO_SYNC  // serialized access to shared data;
    if (threadIdx.x == 0) atomicAddF(out, cache[0], mutex);
#else  // no sync, what most likely happens is:
       // 1) all threads read 0
       // 2) all threads write concurrently 16 (local block dot product)
    if (threadIdx.x == 0) *out += cache[0];
#endif
}

// cpu implementation of dot product
real_t dot(const real_t* v1, const real_t* v2, int N) {
    real_t s = 0;
    for (int i = 0; i != N; ++i) {
        s += v1[i] * v2[i];
    }
    return s;
}

// initialization function run on the GPU
__global__ void init_vector(real_t* v, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < N) {
        v[i] = 1.0f;  // real_t( i ) / 1000000.f;
        i += gridDim.x * blockDim.x;
    }
}

//------------------------------------------------------------------------------
int main(int argc, char** argv) {
    const size_t ARRAY_SIZE = 1024;  // 1024 * 1024; //1Mi elements
    const int BLOCKS = 64;           // 512;
    const int THREADS_PER_BLOCK =
        BLOCK_SIZE;  // 256; // total threads = 512 x 256 = 128ki threads;
    const size_t SIZE = ARRAY_SIZE * sizeof(real_t);

    // device storage
    real_t* dev_v1 = 0;   // vector 1
    real_t* dev_v2 = 0;   // vector 2
    real_t* dev_out = 0;  // result
    int* dev_mutex = 0;
    hipMalloc(&dev_v1, SIZE);
    hipMalloc(&dev_v2, SIZE);
    hipMalloc(&dev_out, sizeof(real_t));
    hipMalloc(&dev_mutex, sizeof(int));

    // host storage
    std::vector<real_t> host_v1(ARRAY_SIZE);
    std::vector<real_t> host_v2(ARRAY_SIZE);
    real_t host_out = 0.f;

    // initialize vector 1 with kernel; much faster than using for loops on the
    // cpu
    init_vector<<<1024, 256>>>(dev_v1, ARRAY_SIZE);
    hipMemcpy(&host_v1[0], dev_v1, SIZE, hipMemcpyDeviceToHost);
    // initialize vector 2 with kernel; much faster than using for loops on the
    // cpu
    init_vector<<<1024, 256>>>(dev_v2, ARRAY_SIZE);
    hipMemcpy(&host_v2[0], dev_v2, SIZE, hipMemcpyDeviceToHost);

    // initialize result on GPU: note the use of cudaMemset, alternatives are to
    // run a kernel or copy from CPU
    hipMemset(dev_out, 0, sizeof(real_t));
    hipMemset(dev_mutex, 0, sizeof(int));

    // execute kernel
    full_dot<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_v1, dev_v2, dev_out, ARRAY_SIZE,
                                            dev_mutex);
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

    // copy output data from device(gpu) to host(cpu)
    hipMemcpy(&host_out, dev_out, sizeof(real_t), hipMemcpyDeviceToHost);

    // print dot product by summing up the partially reduced vectors
    std::cout << "GPU: " << host_out << std::endl;

    // print dot product on cpu
    std::cout << "CPU: " << dot(&host_v1[0], &host_v2[0], ARRAY_SIZE)
              << std::endl;

    // free memory
    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_out);

    return 0;
}
