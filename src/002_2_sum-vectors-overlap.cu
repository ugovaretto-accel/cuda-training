#include "hip/hip_runtime.h"
// #CUDA Training
//
// #Example 2.2 - sum vectors, overlap communication and computation
//
// #Author Ugo Varetto
//

#include <algorithm>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

typedef float real_t;

// In this case the kernel assumes that the computation was started with enough
// threads to cover the entire domain. This is the preferred solution provided
// there are enough threads to cover the entire domain which might not be the
// case in case of a 1D grid layout (max number of threads = 512 threads per
// block x 65536  blocks = 2^25 = 32 Mi threads)
__global__ void sum_vectors(const real_t* v1, const real_t* v2, real_t* out,
                            size_t num_elements) {
    // compute current thread id
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    // since we assume that num threads >= num element we need to make sure we
    // do not write outside the range of the output buffer
    if (xIndex < num_elements) out[xIndex] = v1[xIndex] + v2[xIndex];
}

//------------------------------------------------------------------------------
int main(int, char**) {
    // first task: verify support for overlap of communication and computation
    hipDeviceProp_t prop = hipDeviceProp_t();
    int currentDevice = -1;
    hipGetDevice(&currentDevice);
    hipGetDeviceProperties(&prop, currentDevice);
    if (!prop.deviceOverlap) {
        std::cout << "Device doesn't handle computation-communication overlap"
                  << std::endl;
        return 1;
    }

    const size_t VECTOR_SIZE = 0x1000000;
    const size_t NUMBER_OF_CHUNKS = 4;
    const size_t VECTOR_CHUNK_SIZE = VECTOR_SIZE / NUMBER_OF_CHUNKS;
    const size_t FULL_BYTE_SIZE = sizeof(real_t) * VECTOR_SIZE;
    const size_t CHUNK_BYTE_SIZE =
        FULL_BYTE_SIZE / NUMBER_OF_CHUNKS;  // total size in bytes
    const int THREADS_PER_BLOCK = 256;      // number of gpu threads per block
    const int NUMBER_OF_STREAMS = 2;

    // block size: the number of threads per block multiplied by the number of
    // blocks must be at least equal to NUMBER_OF_THREADS
    const int NUMBER_OF_BLOCKS =
        (VECTOR_CHUNK_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    // if number of threads is not evenly divisable by the number of threads per
    // block we need an additional block; the above code can be rewritten as if(
    // NUMBER_OF_THREADS % THREADS_PER_BLOCK == 0) BLOCK_SIZE =
    // NUMBER_OF_THREADS / THREADS_PER_BLOCK; else BLOCK_SIZE =
    // NUMBER_OF_THREADS / THREADS_PER_BLOCK + 1

    // host allocated storage; page locked memory required for async/stream
    // operations
    real_t* v1 = 0;
    real_t* v2 = 0;
    real_t* vout = 0;

    // page locked allocation
    hipHostAlloc(&v1, FULL_BYTE_SIZE, hipHostMallocDefault);
    hipHostAlloc(&v2, FULL_BYTE_SIZE, hipHostMallocDefault);
    hipHostAlloc(&vout, FULL_BYTE_SIZE, hipHostMallocDefault);

    // generate constant element
    struct Gen {
        real_t v_;
        Gen(real_t v) : v_(v) {}
        real_t operator()() const { return v_; }
    };

    std::generate(v1, v1 + VECTOR_SIZE, Gen(1.0f));
    std::generate(v2, v2 + VECTOR_SIZE, Gen(2.0f));
    std::generate(vout, vout + VECTOR_SIZE, Gen(0.f));

    // gpu allocated storage: number of arrays == number of streams == 2
    real_t* dev_in00 = 0;  // v1
    real_t* dev_in01 = 0;  // v1
    real_t* dev_in10 = 0;  // v2
    real_t* dev_in11 = 0;  // v2
    real_t* dev_out0 = 0;  // vout
    real_t* dev_out1 = 0;  // vout

    hipMalloc(&dev_in00, CHUNK_BYTE_SIZE);
    hipMalloc(&dev_in01, CHUNK_BYTE_SIZE);
    hipMalloc(&dev_in10, CHUNK_BYTE_SIZE);
    hipMalloc(&dev_in11, CHUNK_BYTE_SIZE);
    hipMalloc(&dev_out0, CHUNK_BYTE_SIZE);
    hipMalloc(&dev_out1, CHUNK_BYTE_SIZE);

    // streams; each streams is associated with a separate execution queue
    hipStream_t stream0 = hipStream_t();
    hipStream_t stream1 = hipStream_t();
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    // events; for timing
    hipEvent_t start = hipEvent_t();
    hipEvent_t stop = hipEvent_t();
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // record start
    hipEventRecord(start, 0);

#if defined(STREAM_NO_OVERLAP)
    // computation (wrong order):
    for (int i = 0; i < VECTOR_SIZE;
         i += NUMBER_OF_STREAMS * VECTOR_CHUNK_SIZE) {
        hipMemcpyAsync(dev_in00, v1 + i, CHUNK_BYTE_SIZE,
                        hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_in10, v2 + i, CHUNK_BYTE_SIZE,
                        hipMemcpyHostToDevice, stream0);
        sum_vectors<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream0>>>(
            dev_in00, dev_in10, dev_out0, VECTOR_CHUNK_SIZE);
        hipMemcpyAsync(vout + i, dev_out0, CHUNK_BYTE_SIZE,
                        hipMemcpyDeviceToHost, stream0);

        hipMemcpyAsync(dev_in01, v1 + i + VECTOR_CHUNK_SIZE, CHUNK_BYTE_SIZE,
                        hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_in11, v2 + i + VECTOR_CHUNK_SIZE, CHUNK_BYTE_SIZE,
                        hipMemcpyHostToDevice, stream1);
        sum_vectors<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream1>>>(
            dev_in01, dev_in11, dev_out1, VECTOR_CHUNK_SIZE);
        hipMemcpyAsync(vout + i + VECTOR_CHUNK_SIZE, dev_out1, CHUNK_BYTE_SIZE,
                        hipMemcpyDeviceToHost, stream1);
    }
#else
    // computation (correct order, interleaved or not makes little difference)
    for (int i = 0; i < VECTOR_SIZE;
         i += NUMBER_OF_STREAMS * VECTOR_CHUNK_SIZE) {
        hipMemcpyAsync(dev_in00, v1 + i, CHUNK_BYTE_SIZE,
                        hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_in01, v1 + i + VECTOR_CHUNK_SIZE, CHUNK_BYTE_SIZE,
                        hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_in10, v2 + i, CHUNK_BYTE_SIZE,
                        hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_in11, v2 + i + VECTOR_CHUNK_SIZE, CHUNK_BYTE_SIZE,
                        hipMemcpyHostToDevice, stream1);
        sum_vectors<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream0>>>(
            dev_in00, dev_in10, dev_out0, VECTOR_CHUNK_SIZE);
        sum_vectors<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream1>>>(
            dev_in01, dev_in11, dev_out1, VECTOR_CHUNK_SIZE);
        hipMemcpyAsync(vout + i, dev_out0, CHUNK_BYTE_SIZE,
                        hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(vout + i + VECTOR_CHUNK_SIZE, dev_out1, CHUNK_BYTE_SIZE,
                        hipMemcpyDeviceToHost, stream1);
    }
#endif

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float e = float();
    hipEventElapsedTime(&e, start, stop);
    std::cout << "elapsed time (ms): " << e << std::endl;
    // print first and last element of vector
    std::cout << "result: " << vout[0] << ".." << vout[VECTOR_SIZE - 1]
              << std::endl;

    // free memory
    hipFree(dev_in00);
    hipFree(dev_in01);
    hipFree(dev_in10);
    hipFree(dev_in11);
    hipFree(dev_out0);
    hipFree(dev_out1);
    hipHostFree(v1);
    hipHostFree(v2);
    hipHostFree(vout);
    // release streams
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    // release events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
