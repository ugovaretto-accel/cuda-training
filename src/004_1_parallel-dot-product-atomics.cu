// #CUDA Training
//
// #Example 4.1 - dot product with atomics, full dot product on GPU
//
// #Author: Ugo Varetto
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

typedef float real_t;

const size_t BLOCK_SIZE = 16;

// Reduce at each step by summing in place element[i] + element[i+step]
// where 'step'starts at half array length and gets divided by two at each step
//   _______
//  v       v
// 1 1 1 1 1 1 1 1
//^-------^
//
// 2 2 2 2
// 4 4
// 8 <-- add value to global result
// global result variable contained fully computed dot product
// launch grid size <= array size
// number of returned partial redutions == number of blocks
__global__ void full_dot(const real_t* v1, const real_t* v2, real_t* out,
                         int N) {
    __shared__ real_t cache[BLOCK_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cache[threadIdx.x] = 0.f;
    while (i < N) {
        cache[threadIdx.x] += v1[i] * v2[i];
        i += gridDim.x * blockDim.x;
    }
    __syncthreads();  // required because later on the current thread is
                      // accessing data written by another thread
    i = BLOCK_SIZE / 2;
    while (i > 0) {
        if (threadIdx.x < i) cache[threadIdx.x] += cache[threadIdx.x + i];
        __syncthreads();
        i /= 2;  // not sure bitwise operations are actually faster
    }
#ifndef NO_SYNC  // serialized access to shared data;
    if (threadIdx.x == 0) atomicAdd(out, cache[0]);
#else  // no sync, what most likely happens is:
       // 1) all threads read 0
       // 2) all threads write concurrently 16 (local block dot product)
    if (threadIdx.x == 0) *out += cache[0];
#endif
}

real_t dot(const real_t* v1, const real_t* v2, int N) {
    real_t s = 0;
    for (int i = 0; i != N; ++i) {
        s += v1[i] * v2[i];
    }
    return s;
}

__global__ void init_vector(real_t* v, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < N) {
        v[i] = 1.0f;  // real_t( i ) / 1000000.f;
        i += gridDim.x * blockDim.x;
    }
}

//------------------------------------------------------------------------------
int main(int argc, char** argv) {
    const size_t ARRAY_SIZE = 1024;  // 1024 * 1024; //1Mi elements
    const int BLOCKS = 64;           // 512;
    const int THREADS_PER_BLOCK =
        BLOCK_SIZE;  // 256; // total threads = 512 x 256 = 128ki threads;
    const size_t SIZE = ARRAY_SIZE * sizeof(real_t);

    // device storage
    real_t* dev_v1 = 0;   // vector 1
    real_t* dev_v2 = 0;   // vector 2
    real_t* dev_out = 0;  // result
    hipMalloc(&dev_v1, SIZE);
    hipMalloc(&dev_v2, SIZE);
    hipMalloc(&dev_out, sizeof(real_t));

    // host storage
    std::vector<real_t> host_v1(ARRAY_SIZE);
    std::vector<real_t> host_v2(ARRAY_SIZE);
    real_t host_out = 0.f;

    // initialize vector 1 with kernel; much faster than using for loops on the
    // cpu
    init_vector<<<1024, 256>>>(dev_v1, ARRAY_SIZE);
    hipMemcpy(&host_v1[0], dev_v1, SIZE, hipMemcpyDeviceToHost);
    // initialize vector 2 with kernel; much faster than using for loops on the
    // cpu
    init_vector<<<1024, 256>>>(dev_v2, ARRAY_SIZE);
    hipMemcpy(&host_v2[0], dev_v2, SIZE, hipMemcpyDeviceToHost);

    // initialize result on GPU: note the use of cudaMemset, alternatives are to
    // run a kernel or copy from CPU
    hipMemset(dev_out, 0, sizeof(real_t));

    // execute kernel
    full_dot<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_v1, dev_v2, dev_out,
                                            ARRAY_SIZE);
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

    // copy output data from device(gpu) to host(cpu)
    hipMemcpy(&host_out, dev_out, sizeof(real_t), hipMemcpyDeviceToHost);

    // print dot product by summing up the partially reduced vectors
    std::cout << "GPU: " << host_out << std::endl;

    // print dot product on cpu
    std::cout << "CPU: " << dot(&host_v1[0], &host_v2[0], ARRAY_SIZE)
              << std::endl;

    // free memory
    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_out);

    return 0;
}
