// #CSCS CUDA Training 
//
// #Example 7 - constant memory
//
// #Author Ugo Varetto
//
// #Goal:  multiply input array elements by weights and write result into output array;
//         investigate the use of constant memory for storing the weights and how
//         having each thread in each warp access the same element compares
//         to have each thread access a different element from a performance standpoint;
//         also compare the result with storing data in global memory      
//
// #Rationale: constant memory can be used to store a small (64kiB) dataset frequently
//             accessed from a kernel; behavior of constant memory is the opposite of
//             global memory: it is faster to have multiple threads access the same
//             element than having multiple threads access separate elements
//
// #Solution: implement and time different kernels:
//            1) all the threads in a warp access the same element in the const array
//            2) each thread in the grid accesses a different element in the const array;
//            redo the same for global memory timing the computation with events
//
// #Code: flow:
//        1) compute launch grid configuration
//        2) allocate data on host(cpu) and device(gpu)
//        3) copy data from host to device, in this case also copy to const global array on GPU
//        4) launch and time kernels
//        6) synchronize events to wait for end of execution 
//        7) consume data (in this case print result and time)
//        8) free memory and events (used to time operations)
//        
// #Compilation: nvcc -arch=sm_13 7_constant-memory.cu -o constant-memory
//
// #Execution: ./constant-memory
//
// #Note: note how on arch 1.3 devices broadcast access is faster than parallel access in constant
//        memory (quite the opposite in global memory) because access to constant memory is serialized
//        within half-warp: each half-warp can only access one element of const memory at a time     
//       
// #Note: kernel invocations ( foo<<<...>>>(...) ) are *always* asynchronous and a call to 
//        cudaThreadSynchronize() is required to wait for the end of kernel execution from
//        a host thread; in case of synchronous copy operations like cudaMemcpy(...,cudaDeviceToHost)
//        kernel execution is guaranteed to be terminated before data are copied
//
// #Note: -arch=sm_13 allows the code to run on every card with hw architecture GT200 (gtx 2xx) or better
//
// #Note: -arch=sm_13 is the lowest architecture version that supports double precision



//#include <cuda_runtime.h> // automatically added by nvcc

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <sstream>
#include <algorithm>
#include <iterator>

typedef float real_t;

static const int HALF_WARP = 16;
static const int NUMBER_OF_BLOCKS  = 64;
static const int THREADS_PER_BLOCK = 16 * HALF_WARP;
static const int VECTOR_SIZE = THREADS_PER_BLOCK * NUMBER_OF_BLOCKS;
static const int NUM_WEIGHTS = VECTOR_SIZE;
static const int BYTE_SIZE = sizeof( real_t ) * VECTOR_SIZE;

// number of weights must fit in local memory = 64kiB
// 64 blocks x ( 16 x 16 threads per block ) = 16ki elements x 4 bytes per element = 64kiB   

__constant__ real_t weights[ NUM_WEIGHTS ];


// out[ global thread id ] = in[ global thread id ] x weights[ block offset + half warp id ];
// each thread in the half-warp accesses the same element in the constant weight array
__global__ void weight_mul_broadcast( const real_t* vin, real_t* out ) {
    // compute current thread id
    const int xBlock = blockIdx.x * blockDim.x;
    const int xIndex = xBlock + threadIdx.x;          
    out[ xIndex ] = vin[ xIndex ] * weights[ xBlock + threadIdx.x / HALF_WARP ];
}

// out[ global thread id ] = in[ global thread id ] x weights[ global thread id ];
// each thread  accesses a different weight: each access from half warp
// threads is serialized i.e. it will take 16 separate read operations to fill a group of 16 output
// elements as comparaed to 16 parallel transfers or less(when coalesced) in the case of global
// memory 
__global__ void weight_mul_separate( const real_t* vin, real_t* out ) {
    // compute current thread id
    const int xBlock = blockIdx.x * blockDim.x;
    const int xIndex = xBlock + threadIdx.x;              
    out[ xIndex ] = vin[ xIndex ] * weights[ xBlock + threadIdx.x ];
}


// same as weight_mul_parallel but reading weights from global memory
__global__ void weight_mul_global_separate( const real_t* vin, const real_t* w, real_t* out ) {
    // compute current thread id
    const int xBlock = blockIdx.x * blockDim.x;
    const int xIndex = xBlock + threadIdx.x;              
    out[ xIndex ] = vin[ xIndex ] * w[ xBlock + threadIdx.x ];
}

// same as weight_mul_broadcast but reading weights from global memory
__global__ void weight_mul_global_broadcast( const real_t* vin, const real_t* w, real_t* out ) {
    // compute current thread id
    const int xBlock = blockIdx.x * blockDim.x;
    const int xIndex = xBlock + threadIdx.x;              
    out[ xIndex ] = vin[ xIndex ] * w[  xBlock + threadIdx.x / HALF_WARP ];//+ threadIdx.x / HALF_WARP ];
}

// generate sequence {i,i+1,i+2,...}
struct GenSeq {
    static int v_; 
    GenSeq( real_t v )  { v_ = v; }
    real_t operator()() const { return v_++; }
};
int GenSeq::v_ = 0;


//------------------------------------------------------------------------------
int main( int , char**  ) {
         
    // host allocated storage; page locked memory required for async/stream operations
    std::vector< real_t > v( VECTOR_SIZE, 1.f );
    std::vector< real_t > host_w( NUM_WEIGHTS );
    std::vector< real_t > vout( VECTOR_SIZE );
   
    std::generate( host_w.begin(), host_w.end(), GenSeq( 0.0f ) );
    std::cout << "Input: ";
    std::copy( v.begin(), v.begin() + 10, std::ostream_iterator< real_t >( std::cout, ", ") ); 
    std::cout << " ..." << std::endl;
    std::cout << "Weigths: ";
    std::copy( host_w.begin(), host_w.begin() + 10, std::ostream_iterator< real_t >( std::cout, ", ") ); 
    std::cout << " ..." << std::endl;
 
    //upload data to const global on GPU (data are const on the GPU, must be initialized from the CPU)
    hipMemcpyToSymbol( HIP_SYMBOL(weights), &host_w[ 0 ], sizeof( real_t ) * NUM_WEIGHTS );
 
    // gpu allocated storage
    real_t* dev_vin  = 0;
    real_t* dev_vout = 0;
    real_t* dev_w    = 0;

    hipMalloc( &dev_vin,  BYTE_SIZE );
    hipMalloc( &dev_vout, BYTE_SIZE );
    hipMalloc( &dev_w, sizeof( real_t) * NUM_WEIGHTS );

    hipMemcpy( dev_vin, &v[ 0 ], BYTE_SIZE, hipMemcpyHostToDevice );
    
    hipMemcpy( dev_w,   &host_w[ 0 ], sizeof( real_t ) * NUM_WEIGHTS, hipMemcpyHostToDevice );
   
    // events; for timing
    hipEvent_t start = hipEvent_t();
    hipEvent_t stop  = hipEvent_t();
    hipEventCreate( &start );
    hipEventCreate( &stop );

    float e = float();

    hipEventRecord( start, 0 );
    weight_mul_broadcast<<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>( dev_vin, dev_vout );
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );    
    hipEventElapsedTime( &e, start, stop );
    std::cout << "Const broadcast:  " << e << " ms" << std::endl;
    //copy data from GPU and print result
    hipMemcpy( &vout[ 0 ], dev_vout, BYTE_SIZE, hipMemcpyDeviceToHost );
    std::cout << "Result: ";
    std::copy( vout.begin(), vout.begin() + 48, std::ostream_iterator< real_t >( std::cout, ", ") ); 
    std::cout << " ...\n" << std::endl;

    hipEventRecord( start, 0 );
    weight_mul_separate<<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>( dev_vin, dev_vout );
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &e, start, stop );
    std::cout << "Const separate: " << e << " ms" << std::endl;
     //copy data from GPU and print result
    hipMemcpy( &vout[ 0 ], dev_vout, BYTE_SIZE, hipMemcpyDeviceToHost );
    std::cout << "Result: ";
    std::copy( vout.begin(), vout.begin() + 48, std::ostream_iterator< real_t >( std::cout, ", ") ); 
    std::cout << " ...\n" << std::endl;

    hipEventRecord( start, 0 );
    weight_mul_global_broadcast<<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>( dev_vin, dev_w, dev_vout );
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &e, start, stop );
    std::cout << "Global broadcast:     " << e << " ms" << std::endl;
    //copy data from GPU and print result
    hipMemcpy( &vout[ 0 ], dev_vout, BYTE_SIZE, hipMemcpyDeviceToHost );
    std::cout << "Result: ";
    std::copy( vout.begin(), vout.begin() + 48, std::ostream_iterator< real_t >( std::cout, ", ") ); 
    std::cout << " ...\n" << std::endl;
  
    hipEventRecord( start, 0 );
    weight_mul_global_separate<<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>( dev_vin, dev_w, dev_vout );
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &e, start, stop );
    std::cout << "Global separate:     " << e << " ms" << std::endl;
    //copy data from GPU and print result
    hipMemcpy( &vout[ 0 ], dev_vout, BYTE_SIZE, hipMemcpyDeviceToHost );
    std::cout << "Result: ";
    std::copy( vout.begin(), vout.begin() + 48, std::ostream_iterator< real_t >( std::cout, ", ") ); 
    std::cout << " ...\n" << std::endl;

    // free memory
    hipFree( dev_vin  );
    hipFree( dev_w    );
    hipFree( dev_vout );

    // release events
    hipEventDestroy( start );
    hipEventDestroy( stop  );

    return 0;
}
